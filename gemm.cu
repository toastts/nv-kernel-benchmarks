#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <runner.cuh>
#include <vector>

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

const std::string errLogFile = "matrixValidationFailure.txt";

const char *kernel_names[] = {"cuBLAS FP32", "Naive GEMM",
                              "Global Memory Coalescing GEMM"};

int main(int argc, char **argv) {
  if (argc != 2) {
    std::cerr << "select kernel range from [0,2]" << std::endl;
    exit(EXIT_FAILURE);
  }

  // get kernel number
  int kernel_num = std::stoi(argv[1]);
  if (kernel_num < 0 || kernel_num > 4) {
    std::cerr << "Please enter a valid kernel number [0,4]" << std::endl;
    exit(EXIT_FAILURE);
  }

  // get environment variable for device
  int deviceIdx = 0;
  if (getenv("DEVICE") != NULL) {
    deviceIdx = atoi(getenv("DEVICE"));
  }
  cudaCheck(hipSetDevice(deviceIdx));

  printf("Running kernel %s on device %d.\n", kernel_names[kernel_num],
         deviceIdx);

  // print some device info
  CudaDeviceInfo();

  // Declare the handle, create the handle, hipblasCreate will return a value of
  // type hipblasStatus_t to determine whether the handle was created
  // successfully (the value is 0)
  hipblasHandle_t handle;
  if (hipblasCreate(&handle)) {
    std::cerr << "Create cublas handle error." << std::endl;
    exit(EXIT_FAILURE);
  };

  // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
  // publishing event tasks in the target stream
  float elapsed_time;
  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);

  // cuBLAS FLOPs ceiling is reached at 8192
  std::vector<int> SIZE = {128, 256, 512, 1024, 2048, 4096};

  long m, n, k, max_size;
  max_size = SIZE[SIZE.size() - 1];
  std::cout << "Max size: " << max_size << std::endl;

  float alpha = 1.0, beta = 0.0; // GEMM input parameters, C=α*AB+β*C

  float *A = nullptr, *B = nullptr, *C = nullptr,
        *C_ref = nullptr; // host matrices
  float *dA = nullptr, *dB = nullptr, *dC = nullptr,
        *dC_ref = nullptr; // device matrices

  A = (float *)malloc(sizeof(float) * max_size * max_size);
  B = (float *)malloc(sizeof(float) * max_size * max_size);
  C = (float *)malloc(sizeof(float) * max_size * max_size);
  C_ref = (float *)malloc(sizeof(float) * max_size * max_size);

  randomize_matrix(A, max_size * max_size);
  randomize_matrix(B, max_size * max_size);
  randomize_matrix(C, max_size * max_size);

  cudaCheck(hipMalloc((void **)&dA, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dB, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dC, sizeof(float) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dC_ref, sizeof(float) * max_size * max_size));

  cudaCheck(hipMemcpy(dA, A, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dB, B, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dC, C, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dC_ref, C, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));

  int repeat_times = 50;
  for (int size : SIZE) {
    m = n = k = size;

    std::cout << "dimensions(m=n=k) " << m << ", alpha: " << alpha
              << ", beta: " << beta << std::endl;
    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (kernel_num != 0) {
      run_kernel(0, m, n, k, alpha, dA, dB, beta, dC_ref,
                 handle); // cuBLAS
      run_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC,
                 handle); // Executes the kernel, modifies the result matrix
      cudaCheck(hipDeviceSynchronize());
      cudaCheck(hipGetLastError()); // Check for async errors during kernel run
      hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
      hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);

      if (!verify_matrix(C_ref, C, m * n)) {
        std::cout
            << "Failed to pass the correctness verification against NVIDIA "
               "cuBLAS."
            << std::endl;
        if (m <= 128) {
          std::cout << " Logging faulty output into " << errLogFile << "\n";
          std::ofstream fs;
          fs.open(errLogFile);
          fs << "A:\n";
          print_matrix(A, m, n, fs);
          fs << "B:\n";
          print_matrix(B, m, n, fs);
          fs << "C:\n";
          print_matrix(C, m, n, fs);
          fs << "Should:\n";
          print_matrix(C_ref, m, n, fs);
        }
        exit(EXIT_FAILURE);
      }
    }

    hipEventRecord(beg);
    float h2d_time, d2h_time, kernel_time;
    
    // Time H2D transfers
    hipEvent_t h2d_start, h2d_end;
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_end);
    
    hipEventRecord(h2d_start);
    hipMemcpy(dA, A, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * k * n, hipMemcpyHostToDevice);
    hipEventRecord(h2d_end);
    hipEventSynchronize(h2d_end);
    hipEventElapsedTime(&h2d_time, h2d_start, h2d_end);
    
    // Time kernel execution
    hipEvent_t kernel_start, kernel_end;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_end);
    
    hipEventRecord(kernel_start);
    for (int j = 0; j < repeat_times; j++) {
      run_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle);
    }
    hipEventRecord(kernel_end);
    hipEventSynchronize(kernel_end);
    hipEventElapsedTime(&kernel_time, kernel_start, kernel_end);
    
    // Calculate metrics
    long total_bytes = (m * k + k * n + m * n) * sizeof(float);
    float gb_per_sec = (total_bytes * repeat_times) / (1e9 * kernel_time/1000.0);
    long flops = 2 * m * n * k; // Keep original FLOPS calculation
    float flops_per_byte = (float)flops / total_bytes;
    
    printf(
        "Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS, M B/W: (%7.1f) GB/s, AI: (%4.1f) FLOPS/B, H2D: (%4.1f%%), size: (%ld).\n",
        kernel_time / (repeat_times * 1000.0),
        (repeat_times * flops * 1e-9) / (kernel_time/1000.0),
        gb_per_sec,
        flops_per_byte,
        (h2d_time/kernel_time) * 100,
        m);

    // make dC and dC_ref equal again (we modified dC while calling our kernel
    // for benchmarking)
    cudaCheck(hipMemcpy(dC, dC_ref, sizeof(float) * m * n,
                         hipMemcpyDeviceToDevice));
  }

  // Free up CPU and GPU space
  free(A);
  free(B);
  free(C);
  free(C_ref);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dC_ref);
  hipblasDestroy(handle);

  return 0;
};
